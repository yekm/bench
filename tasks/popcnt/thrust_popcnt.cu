
#include <hip/hip_runtime.h>
#ifdef CUDA_FOUND

#include <thrust/device_vector.h>

template <typename T>
struct thrust_popcnt_kernel
{
    __host__ __device__
    T operator()(const T & x) const
    {
        return __popc(x);
    }
};

/* cant use dbg.h with nvcc. removed #include and copied declaration here */
template<typename T>
size_t thrust_popcnt_ex(typename std::vector<T>::const_iterator begin,
                   typename std::vector<T>::const_iterator end);

template<>
size_t thrust_popcnt_ex<int>(typename std::vector<int>::const_iterator begin,
                          typename std::vector<int>::const_iterator end)
{
    thrust::device_vector<int> d_vec(begin, end);
    return thrust::transform_reduce(d_vec.begin(), d_vec.end(),
                                    thrust_popcnt_kernel<int>(),
                                    size_t(), thrust::plus<int>());
}

#endif
