#include "hip/hip_runtime.h"
#include "thrust_popcnt.hpp"

#ifdef CUDA_FOUND

#include <thrust/device_vector.h>

template <typename T>
struct thrust_popcnt
{
    __host__ __device__
    T operator()(const T & x) const
    {
        return __popc(x);
    }
};

template<>
int thrust_popcnt_ex<int>(const std::vector<int> & d)
{
    thrust::device_vector<int> d_vec(d.begin(), d.end());
    return thrust::transform_reduce(d_vec.begin(), d_vec.end(),
                                    thrust_popcnt<int>(), 0, thrust::plus<int>());
}

#endif
