#include "hip/hip_runtime.h"
#include <vector>
#include <typeinfo>

#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>

__device__
unsigned int hash(unsigned int a)
{
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

template<typename T>
class RealRandomNumberFunctor : public thrust::unary_function<unsigned int, T>
{
public:
    RealRandomNumberFunctor(unsigned int seed, T min, T max)
        : m_seed(seed)
        , m_min(min)
        , m_max(max)
    {}

    __device__
    T operator()(unsigned int thread_id)
    {
        thrust::default_random_engine rng(m_seed + hash(thread_id));
        thrust::uniform_real_distribution<T> distribution(m_min, m_max);
        return distribution(rng);
    }

private:
    unsigned int m_seed;
    T m_min, m_max;
};

template<typename T>
class IntRandomNumberFunctor : public thrust::unary_function<unsigned int, T>
{
public:
    IntRandomNumberFunctor(unsigned int seed, T min, T max)
        : m_seed(seed)
        , m_min(min)
        , m_max(max)
    {}

    __device__
    T operator()(unsigned int thread_id)
    {
        thrust::default_random_engine rng(m_seed + hash(thread_id));
        thrust::uniform_int_distribution<T> distribution(m_min, m_max);
        return distribution(rng);
    }

private:
    unsigned int m_seed;
    T m_min, m_max;
};

template<typename T>
void thrust_generate_random_ex(typename std::vector<T>::iterator,
                               size_t, unsigned int, T, T);
/*
{
    std::string message("thrust random generator do not support ");
    throw std::logic_error(message + typeid(T).name());
    // and then I realised that it will be never used.
}
*/

template<>
void thrust_generate_random_ex<float>(std::vector<float>::iterator begin,
                                   size_t size,
                                   unsigned int seed,
                                   float min, float max)
{
    thrust::device_vector<float> d_vec(size);
    thrust::transform(thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(size),
            d_vec.begin(), RealRandomNumberFunctor<float>(seed, min, max));

    thrust::copy(d_vec.begin(), d_vec.end(), begin);
}

template<>
void thrust_generate_random_ex<int>(std::vector<int>::iterator begin,
                                 size_t size,
                                 unsigned int seed,
                                 int min, int max)
{
    thrust::device_vector<int> d_vec(size);
    thrust::transform(thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(size),
            d_vec.begin(), IntRandomNumberFunctor<int>(seed, min, max));

    thrust::copy(d_vec.begin(), d_vec.end(), begin);
}

template<>
void thrust_generate_random_ex<char>(std::vector<char>::iterator begin,
                                  size_t size,
                                  unsigned int seed,
                                  char min, char max)
{
    thrust::device_vector<char> d_vec(size);
    thrust::transform(thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(size),
            d_vec.begin(), IntRandomNumberFunctor<char>(seed, min, max));

    thrust::copy(d_vec.begin(), d_vec.end(), begin);
}

size_t cuda_get_free_mem()
{
    size_t mem_tot;
    size_t mem_free;
    hipMemGetInfo(&mem_free, &mem_tot);
    return mem_free;
    //std::cout << "Free memory : " << mem_free << std::endl;
    //std::cout << "Total memory : " << mem_tot << std::endl;
}
