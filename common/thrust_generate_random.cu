#include "hip/hip_runtime.h"
#include <vector>
#include <typeinfo>

#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>

__host__ __device__
unsigned int hash(unsigned int a)
{
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

template<typename T>
class RealRandomNumberFunctor : public thrust::unary_function<unsigned int, T>
{
public:
    RealRandomNumberFunctor(unsigned int seed, T min, T max)
        : m_seed(seed)
        , m_min(min)
        , m_max(max)
    {}

    __host__ __device__
        float operator()(unsigned int thread_id)
    {
        thrust::default_random_engine rng(m_seed + hash(thread_id));
        thrust::uniform_real_distribution<T> urd(m_min, m_max);
        return urd(rng);
    }

private:
    unsigned int m_seed;
    T m_min, m_max;
};

template<typename T>
class IntRandomNumberFunctor : public thrust::unary_function<unsigned int, T>
{
public:
    IntRandomNumberFunctor(unsigned int seed, T min, T max)
        : m_seed(seed)
        , m_min(min)
        , m_max(max)
    {}

    __host__ __device__
        float operator()(unsigned int thread_id)
    {
        thrust::default_random_engine rng(m_seed + hash(thread_id));
        thrust::uniform_int_distribution<T> urd(m_min, m_max);
        return urd(rng);
    }

private:
    unsigned int m_seed;
    T m_min, m_max;
};

template<typename T>
void thrust_generate_random(std::vector<T> &, unsigned int, T, T);
/*
{
    std::string message("thrust random generator do not support ");
    throw std::logic_error(message + typeid(T).name());
    // and then I realised that it will be never used.
}
*/

template<>
void thrust_generate_random<float>(std::vector<float> & v, unsigned int seed, float min, float max)
{
    thrust::device_vector<float> d_vec(v.size());
    thrust::transform(thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(v.size()),
            d_vec.begin(), RealRandomNumberFunctor<float>(seed, min, max));

    thrust::copy(d_vec.begin(), d_vec.end(), v.begin());
}

template<>
void thrust_generate_random<int>(std::vector<int> & v, unsigned int seed, int min, int max)
{
    thrust::device_vector<int> d_vec(v.size());
    thrust::transform(thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(v.size()),
            d_vec.begin(), IntRandomNumberFunctor<int>(seed, min, max));

    thrust::copy(d_vec.begin(), d_vec.end(), v.begin());
}

template<>
void thrust_generate_random<char>(std::vector<char> & v, unsigned int seed, char min, char max)
{
    thrust::device_vector<char> d_vec(v.size());
    thrust::transform(thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(v.size()),
            d_vec.begin(), IntRandomNumberFunctor<char>(seed, min, max));

    thrust::copy(d_vec.begin(), d_vec.end(), v.begin());
}
